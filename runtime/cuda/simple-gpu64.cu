#include "hip/hip_runtime.h"

__device__ int ave(int a, int b)
{
   return (a+b)/2;
}

extern "C" __global__ void simple(int *data)
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   data[tid] = ave(tid, tid);
}

texture<int, hipTextureType1D, hipReadModeElementType> tex;
extern "C" __global__ void simple_tex(int *data)
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   data[tid] = tex1Dfetch(tex, tid);
}

